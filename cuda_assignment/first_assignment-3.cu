#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <mpi.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>


#define N 10


// function to randomly initialize matrices
void random_mat(double* mat, int mat_size, unsigned int seed) {

    // set seed
    srand(seed);

    // set factor to obtain elements with at most an order of 
    // magnitude ~10^6 (to avoid overflow)
    double exp = (6. - log10((double) mat_size)) / 2.;
    double factor = pow(10., exp);

    for (int i=0; i<mat_size; i++)
        mat[i] = (double) rand() / (double) RAND_MAX * factor;
}

// function to create blocks to send to other processes
// 
// 'offset' is the first element of block and 'jump' is the distance between
// first elements of each row of the block
void create_block(double* mat, double* block, int block_y, int block_x, int offset, int jump) {

    // copy data in block
    for (int row=0; row<block_y; row++) {
        int row_offset = offset + row*jump;
        
        for (int i=0; i<block_x; i++)
            block[row*block_x + i] = mat[row_offset + i];
    }
}


int main(int argc, char** argv) {

    int my_rank, n_procs;

    // init MPI
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &my_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &n_procs);

    // init cublas handle
    hipblasHandle_t cublas_handle;
    hipblasStatus_t status = hipblasCreate(&cublas_handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS initialization failed\n");
        return 1;
    }

    // set devices 
    int n_devices;
    hipError_t check_setdev = hipGetDeviceCount(&n_devices);
    if (check_setdev != hipSuccess)
        fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(check_setdev));
    ////////////////////////////////////////////////////////////
    printf("%d sees %d devices\n", my_rank, n_devices);
    ////////////////////////////////////////////////////////////
    hipSetDevice(my_rank % n_devices);

    // compute local matrices size
    int N_loc_short = N / n_procs;
    int N_loc_long = N_loc_short + 1;
    int N_rest = N % n_procs;
    int N_loc;
    if (my_rank < N_rest)
        N_loc = N_loc_long;
    else
        N_loc = N_loc_short;
    
    // define array to store sizes pf blocks to be received
    int* counts_recv = (int*) malloc(n_procs * sizeof(int));
    for (int count=0; count<N_rest; count++)
        counts_recv[count] = N_loc_long*N_loc_long;
    for (int count=N_rest; count<n_procs; count++) {
        if (N_rest)
            counts_recv[count] = N_loc_short*N_loc_long;
        else
            counts_recv[count] = N_loc_short*N_loc_short;
    }

    // define array with positions of blocks to be received
    int* displacements = (int*) malloc(n_procs * sizeof(int));
    displacements[0] = 0;
    int while_count = 1;
    while (while_count < n_procs) {
        displacements[while_count] = displacements[while_count-1] + counts_recv[while_count-1];
        while_count++;
    }

    // allocate local matrices
    double* A = (double*) malloc(N_loc * N * sizeof(double));
    double* B = (double*) malloc(N_loc * N * sizeof(double));
    double* C = (double*) malloc(N_loc * N * sizeof(double));

    // initialize A and B with personal seeds
    double current_time = MPI_Wtime();
    unsigned int my_seed = (unsigned int) (current_time + my_rank + 1);  // '+1' needed because seeds 0 and 1 give same results
    random_mat(A, N_loc*N, my_seed);
    my_seed += n_procs;
    random_mat(B, N_loc*N, my_seed);
    
    // allocate needed local matrices on device and copy data
    double* A_dev;
    double* C_dev;
    hipMalloc((void**) &A_dev, N_loc * N * sizeof(double));
    hipMemcpy(A_dev, A, N_loc * N * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc((void**) &C_dev, N_loc * N * sizeof(double));
    
    ///////////////////////////////////////////////////////////////////////////
    //for (int i=0; i<N_loc*N; i++)
    //    C[i] = 0.1;
    //hipMemcpy(C_dev, C, N_loc*N*sizeof(double), hipMemcpyHostToDevice);
    ///////////////////////////////////////////////////////////////////////////
    
    ///////////////////////////////////////////////////////////////////////////
    //printf("I'm %d of %d\n", my_rank, n_procs);
    //MPI_Barrier(MPI_COMM_WORLD);
    //////////////////////////////////////////////////////////////////////////

    //////////////////////////////////////////////////////////////
    if (my_rank == 0) {
        FILE* file = fopen("A3.bin", "wb");
        fwrite(A, sizeof(double), N_loc*N, file);
        fclose(file);
        file = fopen("B3.bin", "wb");
        fwrite(B, sizeof(double), N_loc*N, file);
        fclose(file);
    }
    MPI_Barrier(MPI_COMM_WORLD);
    for (int count=1; count<n_procs; count++) {
        if (my_rank == count) {
            FILE* file = fopen("A3.bin", "ab");
            fwrite(A, sizeof(double), N_loc*N, file);
            fclose(file);
            file = fopen("B3.bin", "ab");
            fwrite(B, sizeof(double), N_loc*N, file);
            fclose(file);
        }
        MPI_Barrier(MPI_COMM_WORLD);
    }
    //////////////////////////////////////////////////////////////

    // define quantities for blocks computation
    int offset = 0;  // offset of C blocks
    int N_rows = N_loc;  // just rename the variable for clarity
    int N_cols = N_loc_long;

    // allocate auxiliary matrices
    double* B_block = (double*) malloc(N_rows * N_cols * sizeof(double));  // matrix to store process's block
    double* B_col = (double*) malloc(N * N_cols * sizeof(double));  // matrix to store received blocks

    for (int count=0; count<n_procs; count++) {
	
        if (count == N_rest) {
            // update number of columns and reallocate auxiliary matrices
            N_cols = N_loc_short;
            B_block = (double*) realloc(B_block, N_rows * N_cols * sizeof(double));
            B_col = (double*) realloc(B_col, N * N_cols * sizeof(double));

            // update count_recv and displacements arrays
            for (int count2=0; count2<N_rest; count2++)
                counts_recv[count2] = N_loc_long*N_loc_short;
            for (int count2=N_rest; count2<n_procs; count2++)
                counts_recv[count2] = N_loc_short*N_loc_short;  // not changed in case of zero rest
            while_count = 1;
            while (while_count < n_procs) {
                displacements[while_count] = displacements[while_count-1] + counts_recv[while_count-1];
                while_count++;
            }
       }

        // allocate auxiliary matrices on device
        double* B_col_dev;
        hipError_t check_malloc = hipMalloc((void**) &B_col_dev, N * N_cols * sizeof(double));
	if (check_malloc != hipSuccess)
	    fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(check_malloc));

        // create block to send to other processes
        create_block(B, B_block, N_rows, N_cols, offset, N);

        // send and receive blocks
        MPI_Allgatherv(B_block, N_rows*N_cols, MPI_DOUBLE, B_col, counts_recv, displacements, MPI_DOUBLE, MPI_COMM_WORLD);

        // copy gathered data to device
        hipError_t check_copy = hipMemcpy(B_col_dev, B_col, N*N_cols*sizeof(double), hipMemcpyHostToDevice);
	if (check_copy != hipSuccess)
	    fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(check_copy));

	///////////////////////////////////////////////////////////////////////////
	//printf("I'm %d of %d\n", my_rank, n_procs);
	//MPI_Barrier(MPI_COMM_WORLD);
	//printf("I'm %d and I have %dx%d\n", my_rank, N_rows, N_cols);
	//MPI_Barrier(MPI_COMM_WORLD);
	//////////////////////////////////////////////////////////////////////////

        // matmul
        // (since hipblasDgemm() works in col-major order, to avoid transpositions we 
        // compute B_col.transpose * A.transpose)
	const double alpha = 1.0;
	const double beta = 0.0;
        hipblasStatus_t check_cublas = hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N_cols, N_rows, N, &alpha, B_col_dev, N_cols, A_dev, N, &beta, &C_dev[offset], N);
	if (check_cublas != HIPBLAS_STATUS_SUCCESS) 
	    fprintf(stderr, "CUDA error: %d\n", check_cublas);

        // update offset of C blocks
        offset += N_cols;

        hipFree(B_col_dev);

	/////////////////////////////////////////////////////////////////
        //check_copy = hipMemcpy(C, C_dev, N_loc * N * sizeof(double), hipMemcpyDeviceToHost);
    	//if (check_copy != hipSuccess)
        //    fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(check_copy));
        //for (int count=0; count<n_procs; count++) {
	//    if (count == my_rank) {
	//	for (int i=0; i<N_loc; i++) {
	//	    for (int j=0; j<N; j++)
	//	        printf("%f ", C[i*N+j]);
	//	    printf("\n");
	//	}
	//    }
	//    MPI_Barrier(MPI_COMM_WORLD);
        //}
	//if (my_rank == 0)
	//    printf("\n");
	/////////////////////////////////////////////////////////////////
    }

    // copy accumulated computation from device to host
    hipError_t check_copy = hipMemcpy(C, C_dev, N_loc * N * sizeof(double), hipMemcpyDeviceToHost);
    if (check_copy != hipSuccess)
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(check_copy));
    
    //////////////////////////////////////////////////////////////
    if (my_rank == 0) {
        FILE* file = fopen("C3.bin", "wb");
        fwrite(C, sizeof(double), N_loc*N, file);
        fclose(file);
    }
    MPI_Barrier(MPI_COMM_WORLD);
    for (int count=1; count<n_procs; count++) {
        if (my_rank == count) {
            FILE* file = fopen("C3.bin", "ab");
            fwrite(C, sizeof(double), N_loc*N, file);
            fclose(file);
        }
        MPI_Barrier(MPI_COMM_WORLD);
    }
    //////////////////////////////////////////////////////////////

    free(counts_recv);
    free(displacements);
    free(A);
    free(B);
    free(C);
    free(B_block);
    free(B_col);

    hipFree(A_dev);
    hipFree(C_dev);

    hipblasDestroy(cublas_handle);

    MPI_Finalize();

    return 0;
}
